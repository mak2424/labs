
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

#define N 10000000

__global__ void add(int *a, int *b, int *c, int *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int max = a[tid];


    if (tid < N) {
        if (b[tid] > max) {
            max = b[tid];
        }

        if (c[tid] > max) {
            max = c[tid];
        }

        result[tid] = max;
    }
}

void array_print(int *array, int count) {
    for(int i = 0; i < count * 2; i++) {
    if (i < count) {
        cout << "" << array[i] << " ";
    }

    if (i == count) {
        cout << " ";
    }

    if (i > count - 1 && i < count * 2) {
        cout << "" << array[N - (count * 2 - i) - 1] << " ";
    }
    }
    cout << endl;
}

int main(void) 
{
    int a[N], b[N], c[N], result[N];
    int *input_a, *input_b, *input_c, *out_result;

    int numThreadsPerBlock = 1000;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

    hipMalloc((void**)&input_a, N * sizeof(int));
    hipMalloc((void**)&input_b, N * sizeof(int));
    hipMalloc((void**)&input_c, N * sizeof(int));
    hipMalloc((void**)&out_result, N * sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
        c[i] = rand() % 10;
    }

    hipMemcpy(input_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(input_b, b, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(input_c, c, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<numBlocks, numThreadsPerBlock>>>(input_a, input_b, input_c, out_result);

    hipMemcpy(result, out_result, N * sizeof(int), hipMemcpyDeviceToHost);

    cout << "Arrays:" << endl;

    array_print(a, 5);
    array_print(b, 5);
    array_print(c, 5);

    cout << endl << "Max:" << endl;

    array_print(result, 5);

    hipFree(input_a);
    hipFree(input_b);
    hipFree(input_c);
    hipFree(out_result);

    return 0;
}