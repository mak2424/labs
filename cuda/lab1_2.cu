#include <iostream>
using namespace std;

int main () {
  int device_count;
  hipGetDeviceCount(&device_count);

  hipDeviceProp_t dp;

  cout << "CUDA device count: " << device_count << endl;

  for(int i = 0; i < device_count; i++) {
    hipGetDeviceProperties(&dp, i);

    cout << i << ": " << dp.name << " with CUDA compute compatibility " << dp.major << "." << dp.minor << endl;
    cout << i << ": Тактовая частота ядра = " << dp.clockRate << endl;

    cout << "Память" << endl;
    cout << i << ": Общий объем графической памяти = " << dp.totalGlobalMem / 1024 / 1024 << endl;
    cout << i << ": Объем памяти констант = " << dp.totalConstMem << endl;
    cout << i << ": Максимальный шаг = " << dp.memPitch << endl;

    cout << "Мультипроцессоры" << endl;
    cout << i << ": Число потоковых мультипроцессоров = " << dp.multiProcessorCount << endl;
    cout << i << ": Объем разделяемой памяти в пределах блока = " << dp.sharedMemPerBlock << endl;
    cout << i << ": Число регистров в пределах блока = " << dp.regsPerBlock << endl;
    cout << i << ": Размер WARP’а (нитей в варпе) = " << dp.warpSize << endl;
    cout << i << ": Максимально допустимое число нитей в блоке = " << dp.maxThreadsPerBlock << endl;
    cout << i << ": Mаксимальную размерность при конфигурации нитей в блоке = " << dp.maxThreadsDim[0] << " " << dp.maxThreadsDim[1] << " " << dp.maxThreadsDim[2] << endl;
    cout << i << ": Максимальную размерность при конфигурации блоков в сетке = " << dp.maxGridSize[0] << " " << dp.maxGridSize[1] << " " << dp.maxGridSize[2] << endl;
  }

  return 0;
}