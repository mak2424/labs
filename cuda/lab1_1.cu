
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void kernel() {
  printf("Hello World!\n");
}

int main () {
  kernel<<<1,2>>>();
  kernel<<<3,1>>>();

  printf("Hello from CPU!\n");

  hipDeviceSynchronize();

  return 0;
}