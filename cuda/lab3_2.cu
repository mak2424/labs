
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    cout << hipGetErrorString(err) << " in file '" << file << "' at line " << line << endl;
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define N 10000000

__global__ void add(int *a, int *b, int *c, int *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int max = a[tid];


    if (tid < N) {
        if (b[tid] > max) {
            max = b[tid];
        }

        if (c[tid] > max) {
            max = c[tid];
        }

        result[tid] = max;
    }
}

void array_print(int *array, int count) {
    for(int i = 0; i < count * 2; i++) {
    if (i < count) {
        cout << "" << array[i] << " ";
    }

    if (i == count) {
        cout << " ";
    }

    if (i > count - 1 && i < count * 2) {
        cout << "" << array[N - (count * 2 - i) - 1] << " ";
    }
    }
    cout << endl;
}

int main(void) 
{
    int a[N], b[N], c[N], result[N];
    int *input_a, *input_b, *input_c, *out_result;

    int numThreadsPerBlock = 1000;
    int numBlocks = (N + numThreadsPerBlock - 1) / numThreadsPerBlock;

    HANDLE_ERROR(hipMalloc((void**)&input_a, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&input_b, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&input_c, N * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void**)&out_result, N * sizeof(int)));

    for (int i = 0; i < N; i++) {
        a[i] = rand() % 10;
        b[i] = rand() % 10;
        c[i] = rand() % 10;
    }

    HANDLE_ERROR(hipMemcpy(input_a, a, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(input_b, b, N * sizeof(int), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(input_c, c, N * sizeof(int), hipMemcpyHostToDevice));

    add<<<numBlocks, numThreadsPerBlock>>>(input_a, input_b, input_c, out_result);

    HANDLE_ERROR(hipMemcpy(result, out_result, N * sizeof(int), hipMemcpyDeviceToHost));

    cout << "Arrays:" << endl;

    array_print(a, 5);
    array_print(b, 5);
    array_print(c, 5);

    cout << endl << "Max:" << endl;

    array_print(result, 5);

    hipFree(input_a);
    hipFree(input_b);
    hipFree(input_c);
    hipFree(out_result);

    return 0;
}