
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

static void HandleError(hipError_t err, const char *file, int line) {
  if (err != hipSuccess) {
    cout << hipGetErrorString(err) << " in file '" << file << "' at line " << line << endl;
    exit(EXIT_FAILURE);
  }
}

#define HANDLE_ERROR(err) (HandleError(err, __FILE__, __LINE__))

#define N 10

__global__ void add(int *a, int *b, int *c) {
  int tid = blockIdx.x;

  if(tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

int main(void) {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  HANDLE_ERROR(hipMalloc((void**)&dev_a, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_b, N * sizeof(int)));
  HANDLE_ERROR(hipMalloc((void**)&dev_c, N * sizeof(int))); //Вот тут ошибка


  for (int i = 0; i < N; i++) {
    a[i] = -i;
    b[i] = i * i;
  }

  HANDLE_ERROR(hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice));
  HANDLE_ERROR(hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice));

  add<<<N, 1>>>(dev_a, dev_b, dev_c);

  HANDLE_ERROR(hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost));

  for (int i = 0; i < N; i++) {
    cout << a[i] << " + " << b[i] << " = " << c[i] << endl;
  }

  HANDLE_ERROR(hipFree( dev_a ));
  HANDLE_ERROR(hipFree( dev_b ));
  HANDLE_ERROR(hipFree( dev_c ));

  return 0;
}